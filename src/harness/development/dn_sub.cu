
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <assert.h>
#include <hipblas.h>
#include <hipsolver.h>

#define BLOCK_SIZE 512
#define SWAP 32
/******************************************************************************************
4/25 bsauk
This will be used for best subset selection with dgels as the linear least squares routine.
This method will use a greedy forward approach similiar to the one used by subset.f90.

The point of this is to have a fair comparison between best subset regression with dgels and
with LSQ. 

In this approach, I will solve LLSP varying the number of columns. I will find what the best
first variable is, then add the best second variable and so on repeating until the maximum
variable size.

I will choose to add variables if the sum of squared error is minimized. 
Either we need to copy columns or use tranpose...

*******************************************************************************************/

void dn_dgels(hipblasHandle_t cublasH, hipsolverHandle_t cudenseH, int m, int n, double* dA, int lda, double* dB, double* newErr) {

  double *d_tau;  // linear memory of gpu                                                 
  int *devInfo = NULL; //info in GPU (device copy)                                        
  double *d_work = NULL;                                                                  
  double *dC = NULL; // Intermediate vector on GPU nxnrhs                                
  int lwork = 0;                                                                          
  int info_gpu = 0;                                                                       
  const double alpha = 1;                                                                 
  const double beta = 0;                                                                  
  hipMalloc((void **)&d_tau, sizeof(double)*m);                              
  hipMalloc((void **)&devInfo, sizeof(int));                                 
  hipMalloc((void **)&dC, sizeof(double)*n);                           
  int incx = 1;                                                                           
  int incy = 1;                                                                           
  hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;                                   
  hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;                             

  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T, m, n, &alpha, dA, lda, dB, incx, &beta, dC, incy);
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);                                         
  
  hipsolverDnDgeqrf_bufferSize(cudenseH, m, n, dA, lda, &lwork);
  hipMalloc((void **)&d_work, sizeof(double)*lwork);                         

  cusolver_status = hipsolverDnDgeqrf(cudenseH, m, n, dA, lda, d_tau, d_work, lwork, devInfo);                                                     
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);                                     
  hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);        
  
  cublas_status = hipblasDtrsv(cublasH, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, n, dA, lda, dC, incx);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);                                         
  cublas_status = hipblasDtrsv(cublasH, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, dA, lda, dC, incx);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);  
  
  cublas_status = hipblasDnrm2(cublasH, n, dC, incx, newErr);

}

void dn_forwrd(int m, int n, double* A, double* B, int max_size) {
  
  hipsolverHandle_t cudenseH = NULL;  // defining handle                                 
  hipblasHandle_t cublasH = NULL;                                                          
  hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;                                   
  hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;                             

  cusolver_status = hipsolverDnCreate(&cudenseH);
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);                                     

  cublas_status = hipblasCreate(&cublasH);                                                 
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);                                         
                                                                                          
  hipError_t cudaStat1 = hipSuccess;                                                    
  hipError_t cudaStat2 = hipSuccess;                                                    

  double sserr = 1.0e100; //Chosen to be large
  int ldda = ((m+31)/32)*32;
  double *dA = NULL, *dB = NULL, *dTemp = NULL, *dPA = NULL, *dX = NULL;

  double *hTemp = (double *)malloc(ldda*max_size*sizeof(double));
  double *hPA = (double *)malloc(ldda*max_size*sizeof(double));
  double *X = (double *)malloc(n*sizeof(double));
  double newErr[1]; 
  hipMalloc((void **)&dTemp, ldda*max_size*sizeof(double));
  hipMalloc((void **)&dPA, ldda*max_size*sizeof(double));
  hipMalloc((void **)&dX, ldda*sizeof(double));

  cudaStat1 = hipMalloc((void **)&dA, m*n*sizeof(double));                              
  cudaStat2 = hipMalloc((void **)&dB, m*sizeof(double));                           
  assert(cudaStat1 == hipSuccess);                                                       
  assert(cudaStat2 == hipSuccess);                                                       
  cudaStat1 = hipMemcpy(dA, A, sizeof(double)*m*n, hipMemcpyHostToDevice);
  cudaStat2 = hipMemcpy(dB, B, sizeof(double)*m, hipMemcpyHostToDevice);  
  assert(cudaStat1 == hipSuccess);                                                       
  assert(cudaStat2 == hipSuccess);                                                       


  bool chosen[n];
  for(int i=0; i<n; i++) {
    chosen[i] = false;
  }

  for(int vars=0; vars<max_size; vars++) {
    bool needVar = true;
    int tempChosen = 0;
    for(int col=0; col<n; col++) {
      if(needVar && !chosen[col]) {
	if(vars > 0) hipMemcpy(dPA, dTemp, sizeof(double)*max_size*m, hipMemcpyDeviceToDevice); // Reset dPA to be the dTemp 
	hipMemcpy(dX, dB, sizeof(double)*ldda, hipMemcpyDeviceToDevice);
	hipMemcpy2D(&dPA[vars], sizeof(double), &dA[col], n*sizeof(double), sizeof(double), m, hipMemcpyDeviceToDevice);
	dn_dgels(cublasH, cudenseH, m, vars+1, dPA, ldda, dX, newErr);
	if(newErr[0] < sserr) {
	  tempChosen = col;
	  sserr = newErr[0];
	} 
      }
      if(col==n-1) {
	chosen[tempChosen] = true;
	hipMemcpy2D(&dTemp[vars], sizeof(double), &dA[tempChosen], n*sizeof(double), sizeof(double), m, hipMemcpyDeviceToDevice);
      }
    }   
  }

  hipFree(dA); hipFree(dB); hipFree(dTemp); hipFree(dPA); hipFree(dX);
  free(hTemp); free(hPA); free(X);
  if(cublasH) hipblasDestroy(cublasH);                                                     
  if(cudenseH) hipsolverDnDestroy(cudenseH);                                               

}
