#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define NB 4

static inline int updiv(int n, int d) {
  return (n+d-1)/d;
}

__device__ __inline__ int dmin(int a, int b) {
  if(a>b) {
    return b;
  } else {
    return a;
  }
}

// As of 4:41pm on 5/3 I have been debugging for the simplified case with smaller loops. This is just to make sure the basic idea works.
// I believe that up to nextr is correct currently!

__global__ void includGPU(int rows, int cols, double* dA, double* dY, double* dD, double* dR, double* dRHS, double* dSSERR, double* dWeights, int blocks) {

  __shared__ double dXblock[(NB)*NB];
  __shared__ double sWeights[NB];
  __shared__ double sY[NB];
  __shared__ double sD[NB];
  __shared__ double sRHS[NB];

  const int idx = blockIdx.x*blockDim.x+threadIdx.x; // Maps to rows
  const int jdx = blockIdx.y*blockDim.y+threadIdx.y; // Maps to columns
  double vsmall = 2.225e-307;
  int nextr = 0;
  int offset = dmin(NB, cols-threadIdx.y-blockIdx.y*blockDim.y);
  double w = 0.0, xk = 0.00, di = 0.00, cbar = 0.00, sbar = 0.00, xi = 0.00, tempR = 0.00, RHSi = 0.0, xy = 0.00, yi = 0.00;

  if(idx >= blockDim.x || jdx > cols) return;
  sD[threadIdx.x] = dD[idx];
  sRHS[threadIdx.x] = dRHS[idx];
  
  for(int i=threadIdx.x; i<rows; i+=blockDim.x) { // i<rows
    dXblock[threadIdx.x*blockDim.x+threadIdx.y] = dA[i*cols+jdx];
    if(threadIdx.y == 0) {
      sWeights[threadIdx.x] = dWeights[i];
      sY[threadIdx.x] = dY[i];
    } 
    int rowsLeft = dmin(NB, rows-i*blockDim.x);
    w = sWeights[threadIdx.x];
    yi = sY[threadIdx.x];
    __syncthreads();
    
    for(int j=0; j<cols; j++) { // j < cols
      __syncthreads();
      di = sD[j];
      RHSi = sRHS[j];
      if(fabs(w) < vsmall) {
	dWeights[i] = w;
	dY[i] = yi;
	break;
      }
      tempR = dR[nextr+jdx];
      for(int k=0; k<threadIdx.x+1; k++) { // k < threadIdx.x+1
	__syncthreads();
	if(j < (blockIdx.y+1)*blockDim.y && j > blockIdx.y*blockDim.y || j==0 && jdx < NB) {
	  xi = dXblock[k*blockDim.x+j]; // Have 32 threads repeat work so they don't sit idle and will all return if need to
	} else {
	  xi = dA[k*cols+j]; // dA does not have a column of 1's so this doesn't work. Workaround, if j==0, xi = 1;
	}
	if(jdx > j) {
	  xk = dXblock[k*blockDim.x+threadIdx.y];
	}
	if(fabs(xi) < vsmall) {
	  nextr = nextr+cols-j-1;
	} else {
	  w = sWeights[k];
	  yi = sY[k];
	  cbar = di/(di+w*xi*xi);
	  sbar = w*xi/(di+w*xi*xi);
	  di = di+w*xi*xi;

	  if(jdx > j) {
	    if(idx == k) {
	      dXblock[k*blockDim.x+threadIdx.y] = xk-xi*tempR;
	      dA[k*cols+jdx] = xk-xi*tempR;  // How do we ensure that no thread accesses this data before this value is updated?
	    }
	    tempR = cbar*tempR+sbar*xk;
	    if(k == threadIdx.x) nextr = nextr+cols-j-1;
	  }

	  w = cbar*w;
	  xy = yi;
	  yi = xy-xi*RHSi;
	  RHSi = cbar*RHSi+sbar*xy;
	}
      }
      if(threadIdx.x == rowsLeft-1) {
	sD[threadIdx.y] = di;
	sRHS[threadIdx.y] = RHSi;
      }
    }
    if(jdx == cols) { // Idea here is that we need all threads in the row to have grabbed these values before we update. 
      dWeights[i] = w;
      dY[i] = yi;
    }
  }
  
  // This will move the values stored in the shared state to the global variables, that I will need later!
  dD[jdx] = sD[threadIdx.y];
  dRHS[jdx] = sRHS[threadIdx.y];

  for(int i=threadIdx.x; i<rows; i+=blockDim.x) {
    dA[i*cols+jdx] = dXblock[i*blockDim.x+threadIdx.y];
  }

  if(idx==0 && jdx==0) {
    for(int i=0; i<rows; i++) {
      for(int j=0; j<cols; j++) {
	printf("dA(%d,%d) = %f\n", i,j,dA[i*cols+j]);
      }
    }
  }

  if(jdx==0 && idx==0) {
    for(int i=0; i<rows; i++) {
      dSSERR[0] = dSSERR[0]+dWeights[i]*dY[i]*dY[i];
      //      atomicAdd(dSSERR, dWeights[idx]*dY[idx]*dY[idx]);
      //     printf("dWeights[%d] = %f\n", i, dWeights[i]);
    }
  }
} 
  
void gpu_lsq(double* A, double* weights, double* y, int rows, int cols, int nbest, int max_size, double** ress, int** lopt, double* bound) {
  /*
  int nvar = cols-1, nobs = 0, r_dim = cols*(cols-1)/2, max_cdim = max_size*(max_size+1)/2;
  double sserr[1], rss[cols], rhs[cols], work[cols], tol[cols], D[cols], r[r_dim];
  double sterr[max_size];
  int vorder[cols], row_ptr[cols], ifault[1], list[4], ier[1];

  bool lindep[cols], tol_set[1], rss_set[1];
  double vsmall = 2.225e-307, total_sumsq;
  double eps = 1e-14;
    
  sserr[0] = 0.0;
  tol_set[0] = false;
  rss_set[0] = false;

  for(int i=0; i<cols; i++) {
    vorder[i] = i;
  }
  row_ptr[0] = 0;
  for(int i=1; i<cols-1; i++) {
    row_ptr[i] = row_ptr[i-1]+cols-i; 
  }
  row_ptr[cols-1] = 0;
  */
  int r_dim = cols*(cols-1)/2;
  double* dA = NULL;
  double* dY = NULL;
  double* dD = NULL; // cols
  double* dR = NULL; //r_dim
  double* dRHS = NULL; //cols
  double* dSSERR = NULL; // cols
  double* dWeights = NULL; //rows

  hipMalloc((void **)&dA, rows*(cols+1)*sizeof(double));
  hipMalloc((void **)&dY, rows*sizeof(double));
  hipMalloc((void **)&dD, cols*sizeof(double));
  hipMalloc((void **)&dR, r_dim*sizeof(double));
  hipMalloc((void **)&dRHS, cols*sizeof(double));
  hipMalloc((void **)&dSSERR, sizeof(double));
  hipMalloc((void **)&dWeights, rows*sizeof(double));

  hipMemcpy(dA, A, rows*(cols+1)*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dY, y, rows*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dWeights, weights, rows*sizeof(double), hipMemcpyHostToDevice); // May want to consider just assuming 1 for now if this takes too long :/.
  
  hipMemset(dD, 0.00, cols*sizeof(double));
  hipMemset(dR, 0.00, r_dim*sizeof(double));
  hipMemset(dRHS, 0.00, cols*sizeof(double));
  hipMemset(dSSERR, 0.00, sizeof(double));

  dim3 threadsPerBlock(NB,NB);
  dim3 blocks(updiv(rows, NB), updiv(cols, NB));
  includGPU<<<blocks, threadsPerBlock>>>(rows, cols, dA, dY, dD, dR, dRHS, dSSERR, dWeights, blocks.x); 
  hipDeviceSynchronize();
  /****************************************************************
  // This part gets translated into CUDA device code.
  for(int i=0; i<rows; i++) {
    xrow[0] = 1.0;
    for(int j=1; j<cols; j++) {
      xrow[j] = A[i*cols+j-1];
    }
    includ(weights[i], xrow, y[i], cols, D, r, rhs, sserr);
  }
  *****************************************************************/
  //  std::cout << "sserr = " << sserr[0] << std::endl;
  /*
  nobs = rows;
  sing(lindep, ifault, cols, D, tol_set, r, tol, row_ptr, rhs, sserr, work);
  if(ifault[0] == 0) {
    std::cout << "QR-factorization is not singular" << std::endl;
  } else {
    for(int i=0; i<nvar; i++) {
      if(lindep[i]) 
	std::cout << vorder[i] << " is exactly linearly related to earlier variables" << std::endl;
    }
  }
  ss(cols, sserr, rss, rss_set, D, rhs);
  
  // Set tolerances and test for singularities
  tolset(cols, work, r, tol, tol_set);
  sing(lindep, ier, cols, D, tol_set, r, tol, row_ptr, rhs, sserr, work);
  if(ier[0] != 0) {
    std::cout << ier[0] << " singularities detected in predictor variables" << std::endl;
    std::cout << "These variables are linearly related to earlier ones:" << std::endl;
    for(int i=0; i<cols; i++) {
      if(lindep[i]) {
	for(int j=0; j<nvar; j++) {
	  if(lindep[j]) {
	    std::cout << vorder[j] << std::endl;
	  }
	}
	break;
      }
    }
  }
  // Not sure if these three need to be called again here...
  tolset(cols, work, r, tol, tol_set);
  sing(lindep, ier, cols, D, tol_set, r, tol, row_ptr, rhs, sserr, work);
  ss(cols, sserr, rss, rss_set, D, rhs);
  
  for(int i=0; i<max_size; i++) {
    report(i, rss[i], max_size, bound, nbest, ress, vorder, lopt);
  }
  
  total_sumsq = rss[0];
  int first = 1;
  int last = cols;

  // The next part is that I will need to implement the different subset selection techniques, pick a few
  // Forward selection
  double startForwrd = CycleTimer::currentSeconds();
  forwrd(first, last, ifault, cols, max_size, D, rhs, r, nbest, rss, bound, ress, vorder, lopt, rss_set, sserr, row_ptr, tol);
  double endForwrd = CycleTimer::currentSeconds();
  std::cout << "Forwrd took " << 1000.f*(endForwrd-startForwrd) << std::endl;
  /*
  for(int i=first; i<max_size; i++) {
    std::cout << "Best subsets found of " << i << " variables" << std::endl;
    std::cout << "     R.S.S.          Variable numbers" << std::endl;
    int pos = (i*i+i)/2;
    for(int j=0; j<nbest; j++) {
      std::cout << ress[i][j] << "    ";
      for(int k=pos; k<pos+i+1; k++) {
	std::cout << lopt[j][k] << "   ";
      }
      std::cout << std::endl;
    }
  }
  */
}

