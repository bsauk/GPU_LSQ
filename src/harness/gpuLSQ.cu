#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "lsq.h"
#include "sub.h"

#define NB 1
#define COLUMNS 512

static inline int updiv(int n, int d) {
  return (n+d-1)/d;
}


__device__ __inline__ int dmin(int a, int b) {
  if(a>b) {
    return b;
  } else {
    return a;
  }
}

__global__ void includGPU(int rows, int cols, double* dA, double* dY, double* dD, double* dR, double* dRHS, double* dSSERR, double* dWeights, int r_dim) {

  extern __shared__ double dXblock[]; // Used shared memory based on the number of columns in a row, passed in function call.

  const int idx = blockIdx.x*blockDim.x+threadIdx.x; // Maps to rows
  const int jdx = blockIdx.y*blockDim.y+threadIdx.y; // Maps to columns
  double vsmall = 2.225e-307;
  int perRow = dmin(COLUMNS, cols); 
  double w = 0.0, xk = 0.00, di = 0.00, cbar = 0.00, sbar = 0.00, xi = 0.00, tempR = 0.00, RHSi = 0.0, xy = 0.00, yi = 0.00;
  if(idx >= blockDim.x || jdx >= blockDim.y ) return;
  
  for(int i=threadIdx.x; i<rows; i+=blockDim.x) { // Iterate over all rows
    for(int j=threadIdx.y; j<perRow; j+=blockDim.y) { // Iterate over all columns to get values into shared memory for that row
      dXblock[j] = dA[i*cols+j];
    }
    int nextr = 0;
    w = dWeights[i];
    yi = dY[i];
    
    for(int j=0; j<cols; j++) { //Iterate over all columns
      __syncthreads(); // Ensure that the previous iteration finishes before the next one begins. Possibly too tight of a constraint could relax if next value has finished.
      di = dD[j];  // Diagonal matrix
      RHSi = dRHS[j]; // RHS of equation
      if(fabs(w) < vsmall) { // If the weight is less than 1, go to next row.
	dWeights[i] = w;
	dY[i] = yi;
	break;
      } 
      xi = dXblock[j]; 
      if(fabs(xi) >= vsmall) {
	cbar = di/(di+w*xi*xi);
	sbar = w*xi/(di+w*xi*xi);
	di = di+w*xi*xi;
	for(int colBlock=jdx; colBlock<perRow; colBlock+=blockDim.y) { // This is how I have every thread update a value in the row, and then loop through for all values in the row.
	  if(colBlock > j) { // Only update if a value is larger than the current column.
	    tempR = dR[nextr+colBlock-j-1];
	    xk = dXblock[colBlock];
	    dXblock[colBlock] = xk-xi*tempR;
	    dR[nextr+colBlock-j-1] = cbar*tempR+sbar*xk;
	    tempR = cbar*tempR+sbar*xk;
	  }
	}
	// Update values here
	w = cbar*w;
	xy = yi;
	yi = xy-xi*RHSi;
	RHSi = cbar*RHSi+sbar*xy;
	for(int colBlock=threadIdx.y; colBlock<perRow; colBlock+=blockDim.y) { // This ensures that only one thread that has useful information updates the value in global mem.
	  if(colBlock == j) {
	    dD[colBlock] = di;
	    dRHS[colBlock] = RHSi;
	  }
	}
      }
      nextr = nextr+cols-j-1; // Deals with moving to new position for the R matrix.
    }
    if(jdx==0) {
      dWeights[i] = w;
      dY[i] = yi;
    }
  }
  /*  
  // Used to test accuracy of the parallel code
  if(idx==0 && jdx == 0) {
    for(int i=0; i<r_dim; i++) {
      printf("dR[%d]=%f\n", i, dR[i]);
    }
    for(int i=0; i<cols; i++) {
      printf("D[%d]=%f rhs[%d]=%f\n", i, dD[i], i, dRHS[i]);
    }
  }
  */     
  
  if(jdx==0 && idx==0) { // Have to sequentially add the dSSERR values because atomic_dadd doesn't seem to work in CUDA, even though it is in the documentation.
    for(int i=0; i<rows; i++) {
      dSSERR[0] = dSSERR[0]+dWeights[i]*dY[i]*dY[i];
    }
  }
} 

void gpu_lsq(double* A, double* weights, double* y, int rows, int cols, int nbest, int max_size, double** ress, int** lopt, double* bound, int check) {

  int nvar = cols-1, r_dim = cols*(cols-1)/2;
  double sserr[1], rss[cols], rhs[cols], work[cols], tol[cols], D[cols], r[r_dim];
  int vorder[cols], row_ptr[cols], ifault[1], ier[1];

  bool lindep[cols], tol_set[1], rss_set[1];
    
  sserr[0] = 0.0;
  tol_set[0] = false;
  rss_set[0] = false;

  for(int i=0; i<cols; i++) {
    vorder[i] = i;
  }
  row_ptr[0] = 0;
  for(int i=1; i<cols-1; i++) {
    row_ptr[i] = row_ptr[i-1]+cols-i; 
  }
  row_ptr[cols-1] = 0;

  double startCopy = CycleTimer::currentSeconds();

  double* dA = NULL;
  double* dY = NULL;
  double* dD = NULL; // cols
  double* dR = NULL; //r_dim
  double* dRHS = NULL; //cols
  double* dSSERR = NULL; // cols
  double* dWeights = NULL; //rows
  // Allocate and copy values to the GPU, were not timed.
  hipMalloc((void **)&dA, rows*(cols+1)*sizeof(double));
  hipMalloc((void **)&dY, rows*sizeof(double));
  hipMalloc((void **)&dD, cols*sizeof(double));
  hipMalloc((void **)&dR, r_dim*sizeof(double));
  hipMalloc((void **)&dRHS, cols*sizeof(double));
  hipMalloc((void **)&dSSERR, sizeof(double));
  hipMalloc((void **)&dWeights, rows*sizeof(double));
  double endCopy = CycleTimer::currentSeconds();

  hipMemcpy(dA, A, rows*(cols+1)*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dY, y, rows*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dWeights, weights, rows*sizeof(double), hipMemcpyHostToDevice); // May want to consider just assuming 1 for now if this takes too long.
  
  hipMemset(dD, 0.00, cols*sizeof(double));
  hipMemset(dR, 0.00, r_dim*sizeof(double));
  hipMemset(dRHS, 0.00, cols*sizeof(double));
  hipMemset(dSSERR, 0.00, sizeof(double));

  dim3 threadsPerBlock(NB,updiv(512,NB));
  dim3 blocks(1, 1);
  int shared_size = (cols+1)*NB*sizeof(double);
  hipDeviceSynchronize();
  //  printf("copyTime= %f ms\n", 1000.f*(endCopy-startCopy)); //Used to determine how long it takes to set up the matrix. Not used for comparison
  double startInclud = CycleTimer::currentSeconds();
  includGPU<<<blocks, threadsPerBlock, shared_size>>>(rows, cols, dA, dY, dD, dR, dRHS, dSSERR, dWeights, r_dim);
  hipDeviceSynchronize();
  double endInclud = CycleTimer::currentSeconds();
  printf("%f\n", 1000.f*(endInclud-startInclud));
  // Transfer results back to CPU from GPU!
  hipMemcpy(D, dD, cols*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(r, dR, r_dim*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(rhs, dRHS, cols*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(sserr, dSSERR, sizeof(double), hipMemcpyDeviceToHost);

  sing(lindep, ifault, cols, D, tol_set, r, tol, row_ptr, rhs, sserr, work);
  /*
  if(check) {
    if(ifault[0] == 0) {
      std::cout << "QR-factorization is not singular" << std::endl;
    } else {
      for(int i=0; i<nvar; i++) {
	if(lindep[i]) 
	  std::cout << vorder[i] << " is exactly linearly related to earlier variables" << std::endl;
      }
    }
  }
  */
  ss(cols, sserr, rss, rss_set, D, rhs);
  
  // Set tolerances and test for singularities
  tolset(cols, work, r, tol, tol_set);
  sing(lindep, ier, cols, D, tol_set, r, tol, row_ptr, rhs, sserr, work);
  /*
  if(check) {
    if(ier[0] != 0) {
      std::cout << ier[0] << " singularities detected in predictor variables" << std::endl;
      std::cout << "These variables are linearly related to earlier ones:" << std::endl;
      for(int i=0; i<cols; i++) {
	if(lindep[i]) {
	  for(int j=0; j<nvar; j++) {
	    if(lindep[j]) {
	      std::cout << vorder[j] << std::endl;
	    }
	  }
	  break;
	}
      }
    }
  }
  */
  // Not sure if these three need to be called again here...
  tolset(cols, work, r, tol, tol_set);
  sing(lindep, ier, cols, D, tol_set, r, tol, row_ptr, rhs, sserr, work);
  ss(cols, sserr, rss, rss_set, D, rhs);
  
  for(int i=0; i<max_size; i++) {
    report(i, rss[i], max_size, bound, nbest, ress, vorder, lopt);
  }
  
  int first = 1;
  int last = cols;

  // The next part is that I will need to implement the different subset selection techniques, pick a few
  // Forward selection
  forwrd(first, last, ifault, cols, max_size, D, rhs, r, nbest, rss, bound, ress, vorder, lopt, rss_set, sserr, row_ptr, tol);
  if(check) {
    for(int i=first; i<max_size; i++) {
      std::cout << "Best subsets found of " << i << " variables" << std::endl;
      std::cout << "     R.S.S.          Variable numbers" << std::endl;
      int pos = (i*i+i)/2;
      for(int j=0; j<nbest; j++) {
	std::cout << ress[i][j] << "    ";
	for(int k=pos; k<pos+i+1; k++) {
	  std::cout << lopt[j][k] << "   ";
	}
	std::cout << std::endl;
      }
    }
  }
  // Free allocated memory afterwards.
  hipFree(dA);
  hipFree(dY);
  hipFree(dD);
  hipFree(dR);
  hipFree(dRHS);
  hipFree(dSSERR);
  hipFree(dWeights);
}

